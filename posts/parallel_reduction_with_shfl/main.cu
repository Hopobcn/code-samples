#include "hip/hip_runtime.h"
#include <cstdio>
#include "device_reduce_atomic.h"
#include "device_reduce_block_atomic.h"
#include "device_reduce_warp_atomic.h"
#include "device_reduce_stable.h"
#include "vector_functions.h"
#include "cub/hipcub/hipcub.hpp"

#define cudaCheckError() {                                          \
  hipError_t e=hipGetLastError();                                  \
  if(e!=hipSuccess) {                                               \
  printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));           \
  exit(0); \
  }                                                                  \
}

void RunTest(char* label, void (*fptr)(int* in, int* out, int N), int N, int REPEAT, int* src, int checksum) {
  int *in, *out;
  
  //allocate a buffer that is at least large enough that we can ensure it doesn't just sit in l2.
  int MIN_SIZE=4*1024*1024;
  int size=max(int(sizeof(int)*N),MIN_SIZE);
  
  //compute mod base for picking the correct buffer
  int mod=size/(N*sizeof(int));
  hipEvent_t start,stop;
  hipMalloc(&in,size);
  hipMalloc(&out,sizeof(int)*1024);  //only stable version needs multiple elements, all others only need 1
  hipEventCreate(&start);
  hipEventCreate(&stop);
  cudaCheckError();

  hipMemcpy(in,src,N*sizeof(int),hipMemcpyHostToDevice);
  
  //warm up
  fptr(in,out,N);

  hipDeviceSynchronize();
  cudaCheckError();
  hipEventRecord(start);

  for(int i=0;i<REPEAT;i++) {
    //iterate through different buffers
    int o=i%mod;
    fptr(in+o*N,out,N);
  }
  hipEventRecord(stop);
  hipDeviceSynchronize();
  cudaCheckError();

  float time_ms;
  hipEventElapsedTime(&time_ms,start,stop);
  float time_s=time_ms/(float)1e3;

  float GB=(float)N*sizeof(int)*REPEAT;
  float GBs=GB/time_s/(float)1e9;

  int sum;
  hipMemcpy(&sum,out,sizeof(int),hipMemcpyDeviceToHost);
  cudaCheckError();

  char *valid;
  if(sum==checksum) 
    valid="CORRECT";
  else
    valid="INCORRECT";

  printf("%s: %s, Time: %f s, GB/s: %f\n", label, valid, time_s, GBs); 
  hipEventDestroy(start);
  hipEventDestroy(stop);
  hipFree(in);
  hipFree(out);
  cudaCheckError();
}

void RunTestCub(char* label, int N, int REPEAT, int* src, int checksum) {
  int *in, *out;
  hipEvent_t start,stop;
  
  hipMalloc(&in,sizeof(int)*N);
  hipMalloc(&out,sizeof(int)*1024);  //only stable version needs multiple elements, all others only need 1
  hipEventCreate(&start);
  hipEventCreate(&stop);
  cudaCheckError();

  hipMemcpy(in,src,N*sizeof(int),hipMemcpyHostToDevice);

  size_t temp_storage_bytes;
  int* temp_storage=NULL;
  hipcub::DeviceReduce::Reduce(temp_storage, temp_storage_bytes, in, out, N, hipcub::Sum());
  hipMalloc(&temp_storage,temp_storage_bytes);

  hipDeviceSynchronize();
  cudaCheckError();
  hipEventRecord(start);

  for(int i=0;i<REPEAT;i++) {
    hipcub::DeviceReduce::Reduce(temp_storage, temp_storage_bytes, in, out, N, hipcub::Sum());
  }
  hipEventRecord(stop);
  hipDeviceSynchronize();
  cudaCheckError();

  float time_ms;
  hipEventElapsedTime(&time_ms,start,stop);
  float time_s=time_ms/(float)1e3;

  float GB=(float)N*sizeof(int)*REPEAT;
  float GBs=GB/time_s/(float)1e9;

  int sum;
  hipMemcpy(&sum,out,sizeof(int),hipMemcpyDeviceToHost);
  cudaCheckError();

  char *valid;
  if(sum==checksum) 
    valid="CORRECT";
  else
    valid="INCORRECT";

  printf("%s: %s, Time: %f s, GB/s: %f\n", label, valid, time_s, GBs); 
  hipEventDestroy(start);
  hipEventDestroy(stop);
  hipFree(in);
  hipFree(out);
  hipFree(temp_storage);
  cudaCheckError();
}

int main(int argc, char** argv)
{
  if(argc!=3) {
    printf("Usage: ./reduce num_elems repeat\n");
    exit(0);
  }
  int NUM_ELEMS=atoi(argv[1]);
  int REPEAT=atoi(argv[2]);

  printf("NUM_ELEMS: %d, REPEAT: %d\n", NUM_ELEMS, REPEAT);

  int* vals=(int*)malloc(NUM_ELEMS*sizeof(int));
  int checksum =0;
  for(int i=0;i<NUM_ELEMS;i++) {
    vals[i]=rand()%4;
    checksum+=vals[i];
  }

  RunTest("device_reduce_atomic", device_reduce_atomic,NUM_ELEMS,REPEAT,vals,checksum);
  RunTest("device_reduce_atomic_vector2", device_reduce_atomic_vector2,NUM_ELEMS,REPEAT,vals,checksum);
  //RunTest("device_reduce_atomic_vector4", device_reduce_atomic_vector4,NUM_ELEMS,REPEAT,vals,checksum);
  
  RunTest("device_reduce_warp_atomic",device_reduce_warp_atomic,NUM_ELEMS,REPEAT,vals,checksum);
  RunTest("device_reduce_warp_atomic_vector2",device_reduce_warp_atomic_vector2,NUM_ELEMS,REPEAT,vals,checksum);
  //RunTest("device_reduce_warp_atomic_vector4",device_reduce_warp_atomic_vector4,NUM_ELEMS,REPEAT,vals,checksum);
  
  RunTest("device_reduce_block_atomic",device_reduce_block_atomic,NUM_ELEMS,REPEAT,vals,checksum);
  RunTest("device_reduce_block_atomic_vector2",device_reduce_block_atomic_vector2,NUM_ELEMS,REPEAT,vals,checksum);
  RunTest("device_reduce_block_atomic_vector4",device_reduce_block_atomic_vector4,NUM_ELEMS,REPEAT,vals,checksum);
 
  RunTest("device_reduce_block_atomic_range_loop",device_reduce_block_atomic_range_loop,NUM_ELEMS,REPEAT,vals,checksum);
  RunTest("device_reduce_block_atomic_vector2_range_loop",device_reduce_block_atomic_vector2_range_loop,NUM_ELEMS,REPEAT,vals,checksum);
  RunTest("device_reduce_block_atomic_vector4_range_loop",device_reduce_block_atomic_vector4_range_loop,NUM_ELEMS,REPEAT,vals,checksum);


  RunTest("device_reduce_stable",device_reduce_stable,NUM_ELEMS,REPEAT,vals,checksum);
  RunTest("device_reduce_stable_vector2",device_reduce_stable_vector2,NUM_ELEMS,REPEAT,vals,checksum);
  //RunTest("device_reduce_stable_vector4",device_reduce_stable_vector4,NUM_ELEMS,REPEAT,vals,checksum);

  RunTestCub("device_reduce_cub",NUM_ELEMS,REPEAT,vals,checksum);
  
  free(vals);

}
