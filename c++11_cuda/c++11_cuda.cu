#include "hip/hip_runtime.h"
#include <thrust/device_ptr.h>
#include <thrust/count.h>
#include <thrust/execution_policy.h>

#include <iostream>
#include <sys/stat.h>

/////////////////////////////////////////////////////////////////
// Some utility code to define grid_stride_range
// Normally this would be in a header but it's here
// for didactic purposes. Uses 
#include "range.hpp"
using namespace util::lang;

// type alias to simplify typing...
template<typename T>
using step_range = typename range_proxy<T>::step_range_proxy;

template <typename T>
__device__
step_range<T> grid_stride_range(T begin, T end) {
    begin += blockDim.x * blockIdx.x + threadIdx.x;
    return range(begin, end).step(gridDim.x * blockDim.x);
}
/////////////////////////////////////////////////////////////////

template <typename T, typename Predicate>
__device__ 
void count_if(int *count, T *data, int n, Predicate p)
{ 
  for (auto i : grid_stride_range(0, n)) {
    if (p(data[i])) atomicAdd(count, 1);
  }
}

// Use count_if with a lambda function that searches for x, y, z or w
// Note the use of range-based for loop and initializer_list inside the functor
// We use auto so we don't have to know the type of the functor or array
__global__
void xyzw_frequency(int *count, char *text, int n)
{
  const char letters[] { 'x','y','z','w' };

  count_if(count, text, n, [&](char c) {
    for (const auto x : letters) 
      if (c == x) return true;
    return false;
  });
}

// A bug in CUDA 7.0 causes errors when this is called
// Comment out by default, but will work in CUDA 7.5
#if 0
__global__
void xyzw_frequency_thrust_device(int *count, char *text, int n)
{
  const char letters[] { 'x','y','z','w' };

  *count = thrust::count_if(thrust::device, text, text+n, [&](char c) {
    for (const auto x : letters) 
      if (c == x) return true;
    return false;
  });
}
#endif

// a bug in Thrust 1.8 causes warnings when this is uncommented
// so commented out by default -- fixed in Thrust master branch
#if 0 
void xyzw_frequency_thrust_host(int *count, char *text, int n)
{
  const char letters[] { 'x','y','z','w' };

  *count = thrust::count_if(thrust::host, text, text+n, [&](char c) {
    for (const auto x : letters) 
      if (c == x) return true;
    return false;
  });
}
#endif

int main(int argc, char** argv)
{ 
  const char *filename = "warandpeace.txt";
  struct stat buf;

  stat( filename, &buf );

  int numBytes = buf.st_size;
  std::cout << "File size: " << numBytes << " bytes" << std::endl;
  char *h_text = (char*)malloc(numBytes);

  char *d_text;
  hipMalloc((void**)&d_text, numBytes);
  
  FILE *fp = fopen(filename, "r");
  int len = fread(h_text, sizeof(char), numBytes, fp);
  fclose(fp);
  std::cout << "Read " << len << " byte corpus from " << filename << std::endl;

  hipMemcpy(d_text, h_text, len, hipMemcpyHostToDevice);
  
  int count = 0;
  int *d_count;
  hipMalloc(&d_count, sizeof(int));
  hipMemset(d_count, 0, sizeof(int));

  // Try uncommenting one kernel call at a time
  xyzw_frequency<<<8, 256>>>(d_count, d_text, len);
  //xyzw_frequency_thrust_device<<<1, 1>>>(d_count, d_text, len);
  hipMemcpy(&count, d_count, sizeof(int), hipMemcpyDeviceToHost);
  
  //xyzw_frequency_thrust_host(&count, h_text, len);

  std::cout << "counted " << count << " instances of 'x', 'y', 'z', or 'w' in \"" 
  << filename << "\"" << std::endl;

  hipFree(d_count);
  hipFree(d_text);

  return 0;
}
